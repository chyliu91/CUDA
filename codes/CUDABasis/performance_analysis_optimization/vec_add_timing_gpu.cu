#include "hip/hip_runtime.h"

#include "error.cuh"
#include <cmath>
#include <cstdio>
#include <ctime>

const double EPSILON = 1.0e-5;

__global__ void add(const double *a, const double *b, double *c, int size)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void check(const double *a, const double *b, const double *result, const int N)
{
    for (int i = 0; i < N; i++)
    {
        if (fabs(a[i] + b[i] - result[i]) > EPSILON)
        {
            printf("index %d calculation error\n", i);
        }
    }
}

int main(void)
{
    const int N = 1e8;
    const int M = sizeof(double) * N;
    const int NUM_REPEATS = 10;

    // 申请主机端的内存
    double *h_x = (double *)malloc(M);
    double *h_y = (double *)malloc(M);
    double *h_z = (double *)malloc(M);

    // 产生随机数
    srand((unsigned int)time(NULL));
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = (double)rand() / (double)RAND_MAX;
        h_y[n] = (double)rand() / (double)RAND_MAX;
    }

    // 申请设备端的内存
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);

    // 内存到显存拷贝
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    // 定义执行配置
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;
    printf("grid_size: %d\n", grid_size);

    float time_sum = 0.0;
    for (int i = 0; i < NUM_REPEATS; i++)
    {
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        CUDA_CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // 执行核函数
        add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        float elapsed_time = 0.0;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Round %d, Cost Time: %g ms.\n", i, elapsed_time);
        if (i > 0)
        {
            time_sum += elapsed_time;
        }
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    printf("Average Cost Time: %.3f\n", (float)time_sum / (NUM_REPEATS - 1));

    // 将结果从设备端拷回主机端
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);

    check(h_x, h_y, h_z, N);

    // 释放主机端内存
    free(h_x);
    free(h_y);
    free(h_z);

    // 释放设备端的内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    printf("finished ...... \n");

    return 0;
}
