

#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <ctime>

const double EPSILON = 1.0e-5;

__global__ void add(const double *a, const double *b, double *c, int size);

void check(const double *a, const double *b, const double *result, const int N);

int main(void)
{
    const int N = 1e8 + 1;
    const int M = sizeof(double) * N;

    // 申请主机端的内存
    double *h_x = (double *)malloc(M);
    double *h_y = (double *)malloc(M);
    double *h_z = (double *)malloc(M);

    // 产生随机数
    srand((unsigned int)time(NULL));
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = (double)rand() / (double)RAND_MAX;
        h_y[n] = (double)rand() / (double)RAND_MAX;
    }

    // 申请设备端的内存
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);

    // 内存到显存拷贝
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    // 定义执行配置
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;
    printf("grid_size: %d\n", grid_size);

    // 执行核函数
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    // 将结果从设备端拷回主机端
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);

    check(h_x, h_y, h_z, N);

    // 释放主机端内存
    free(h_x);
    free(h_y);
    free(h_z);

    // 释放设备端的内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    printf("finished ...... \n");
    return 0;
}

__global__ void add(const double *a, const double *b, double *c, int size)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void check(const double *a, const double *b, const double *result, const int N)
{
    for (int i = 0; i < N; i++)
    {
        if (fabs(a[i] + b[i] - result[i]) > 1e-15)
        {
            printf("index %d calculation error\n", i);
        }
    }
}