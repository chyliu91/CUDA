#include "hip/hip_runtime.h"
#include "error.cuh"
#include <cmath>
#include <cstdio>
#include <ctime>

const double EPSILON = 1.0e-5;

__global__ void add(const double *a, const double *b, double *c, int size)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid], b[tid];
    }
}

void check(const double *a, const double *b, const double *result, const int N)
{
    for (int i = 0; i < N; i++)
    {
        if (fabs(a[i] + b[i] - result[i]) > EPSILON)
        {
            printf("index %d calculation error\n", i);
        }
    }
}

int main(void)
{
    const int N = 200;
    const int M = sizeof(double) * N;

    // 申请主机端的内存
    double *h_x = (double *)malloc(M);
    double *h_y = (double *)malloc(M);
    double *h_z = (double *)malloc(M);

    // 产生随机数
    srand((unsigned int)time(NULL));
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = (double)rand() / (double)RAND_MAX;
        h_y[n] = (double)rand() / (double)RAND_MAX;
    }

    // 申请设备端的内存
    double *d_x, *d_y, *d_z;
    CUDA_CHECK(hipMalloc((void **)&d_x, M));
    CUDA_CHECK(hipMalloc((void **)&d_y, M));
    CUDA_CHECK(hipMalloc((void **)&d_z, M));

    // 内存到显存拷贝
    CUDA_CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    // 定义执行配置, 故意将 block_size 设置大于上限 1024 的值
    const int block_size = 1025;
    const int grid_size = (N + block_size - 1) / block_size;

    // 执行核函数
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));

    check(h_x, h_y, h_z, N);

    // 释放主机端内存
    free(h_x);
    free(h_y);
    free(h_z);

    // 释放设备端的内存
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_z));

    printf("finished ...... \n");
    return 0;
}